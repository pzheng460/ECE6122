﻿/*
Author: Peizhen Zheng
Class: ECE6122 (A)
Last Date Modified: Nov 4, 2023

Description:

Implement a CUDA program to simulate a 2D random walk. A random walk is a mathematical process that describes a path consisting of a sequence of random steps. In this assignment, you will simulate a large number of walkers taking steps either north, south, east, or west on a grid, and calculate the average distance they travel from the origin.

*/
#include <iostream>
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

#include <chrono>

// Define aliases for the chrono literals and clock to simplify the code
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::microseconds;

// Function to perform calculations using cudaMalloc
__global__ void calculateWithCudaMalloc(int64_t numWalkers, int64_t numSteps, float* distances, int** states)
{
    int walkerId = blockIdx.x * blockDim.x + threadIdx.x;
    if (walkerId < numWalkers)
    {
        int x = 0;
        int y = 0;
        for (int i = 0; i < numSteps; i++)
        {
            // Simulate a random step in 2D space
            int direction = states[walkerId][i] % 4;
            if (direction == 0) x++;
            else if (direction == 1) x--;
            else if (direction == 2) y++;
            else y--;
        }      
        distances[walkerId] = sqrt(static_cast<float>(x * x + y * y));
    }
}

// Function to perform calculations using cudaMallocHost
__global__ void calculateWithCudaMallocHost(int64_t numWalkers, int64_t numSteps, float* distances, int** states)
{
    int walkerId = blockIdx.x * blockDim.x + threadIdx.x;
    if (walkerId < numWalkers)
    {
        int x = 0;
        int y = 0;
        for (int i = 0; i < numSteps; i++)
        {
            // Simulate a random step in 2D space
            int direction = states[walkerId][i] % 4;
            if (direction == 0) x++;
            else if (direction == 1) x--;
            else if (direction == 2) y++;
            else y--;
        }
        distances[walkerId] = sqrt(static_cast<float>(x * x + y * y));
    }
}

// Function to perform calculations using cudaMallocManaged
__global__ void calculateWithCudaManaged(int64_t numWalkers, int64_t numSteps, float* distances, int** states)
{
    int walkerId = blockIdx.x * blockDim.x + threadIdx.x;
    if (walkerId < numWalkers)
    {
        int x = 0;
        int y = 0;
        for (int i = 0; i < numSteps; i++)
        {
            // Simulate a random step in 2D space
            int direction = states[walkerId][i] % 4;
            if (direction == 0) x++;
            else if (direction == 1) x--;
            else if (direction == 2) y++;
            else y--;
        }
        distances[walkerId] = sqrt(static_cast<float>(x * x + y * y));
    }
}

int main(int argc, char* argv[])
{
    int64_t numWalkers = 1000;  // Default value for numWalkers
    int64_t numSteps = 10000;   // Default value for numSteps

    // if (argc != 5) {
    //     std::cerr << "Usage: " << argv[0] << " -W <Number of Walkers> -I <Number of Steps>" << std::endl;
    //     return 1;
    // }

    for (int i = 1; i < argc; i += 2) {
        std::string flag = argv[i];
        if (flag == "-W") numWalkers = std::stoll(argv[i + 1]);  
        else if (flag == "-I") numSteps = std::stoll(argv[i + 1]); 
    }

    if (numWalkers <= 0 || numSteps <= 0) {
        std::cerr << "Invalid input values." << std::endl;
        return 1;
    }

    // Initialize random seed
    srand(static_cast<unsigned int>(time(nullptr)));

    int** states;
    hipHostMalloc(&states, numWalkers * sizeof(int*), hipHostMallocDefault);
    for (int i = 0; i < numWalkers; i++) {
        hipHostMalloc(&states[i], numSteps * sizeof(int), hipHostMallocDefault);
    }

    // Store the random int
    for (int i = 0; i < numWalkers; i++) {
        for (int j = 0; j < numSteps; j++) {
            states[i][j] = std::rand();  // Generate a random int
        }
    }

    int blockSize = 256;
    int numBlocks = ((numWalkers + blockSize - 1) / blockSize);

    // Start timer for cudaMalloc
    auto startCudaMalloc = high_resolution_clock::now();
    // Allocate memory for distances using different methods
    float* distancesCudaMalloc = nullptr;
    hipMalloc((void**)&distancesCudaMalloc, numWalkers * sizeof(float));
    calculateWithCudaMalloc<<<numBlocks, blockSize>>>(numWalkers, numSteps, distancesCudaMalloc, states);
    hipDeviceSynchronize();  // Wait for the GPU to finish
    // Copy data from GPU
    float* distancesCudaMallocCPU = (float*)malloc(numWalkers * sizeof(float));
    hipMemcpy(distancesCudaMallocCPU, distancesCudaMalloc, numWalkers * sizeof(float), hipMemcpyDeviceToHost);
    // Calculate and print the average distance from the origin
    float avgDistanceCudaMalloc = 0.0f;

    for (int i = 0; i < numWalkers; i++)
    {
        avgDistanceCudaMalloc += distancesCudaMallocCPU[i];
    }

    avgDistanceCudaMalloc /= static_cast<float>(numWalkers);
    // Clean up
    hipFree(distancesCudaMalloc);
    free(distancesCudaMallocCPU);
    auto stopCudaMalloc = high_resolution_clock::now();
    // Calculate the elapsed time in milliseconds
    auto microsecondsCudaMalloc = duration_cast<microseconds>(stopCudaMalloc - startCudaMalloc).count();



    // Start timer for cudaMallocHost
    auto startCudaMallocHost = high_resolution_clock::now();
    // Allocate memory for distances using different methods
    float* distancesCudaMallocHost = nullptr;
    hipHostMalloc((void**)&distancesCudaMallocHost, numWalkers * sizeof(float), hipHostMallocDefault);
    calculateWithCudaMallocHost<<<numBlocks, blockSize>>>(numWalkers, numSteps, distancesCudaMallocHost, states);
    hipDeviceSynchronize();  // Wait for the GPU to finish
    // Calculate and print the average distance from the origin
    float avgDistanceCudaMallocHost = 0.0f;
    for (int i = 0; i < numWalkers; i++)
    {
        avgDistanceCudaMallocHost += distancesCudaMallocHost[i];
    }
    avgDistanceCudaMallocHost /= static_cast<float>(numWalkers);
    // Clean up
    hipHostFree(distancesCudaMallocHost);
    auto stopCudaMallocHost = high_resolution_clock::now();
    // Calculate the elapsed time in milliseconds
    auto microsecondsCudaMallocHost = duration_cast<microseconds>(stopCudaMallocHost - startCudaMallocHost).count();




    // Start timer for cudaMallocManaged
    auto startCudaManaged = high_resolution_clock::now();
    // Allocate memory for distances using different methods
    float* distancesCudaManaged = nullptr;
    hipMallocManaged((void**)&distancesCudaManaged, numWalkers * sizeof(float));
    calculateWithCudaManaged<<<numBlocks, blockSize>>>(numWalkers, numSteps, distancesCudaManaged, states);
    hipDeviceSynchronize();  // Wait for the GPU to finish
    // Copy data from GPU
    float* distancesCudaManagedCPU = (float*)malloc(numWalkers * sizeof(float));
    hipMemcpy(distancesCudaManagedCPU, distancesCudaManaged, numWalkers * sizeof(float), hipMemcpyDeviceToHost);
    // Calculate and print the average distance from the origin
    float avgDistanceCudaManaged = 0.0f;

    for (int i = 0; i < numWalkers; i++)
    {
        avgDistanceCudaManaged += distancesCudaManagedCPU[i];
    }

    avgDistanceCudaManaged /= static_cast<float>(numWalkers);
    // Clean up
    hipFree(distancesCudaManaged);
    free(distancesCudaManagedCPU);
    auto stopCudaManaged = high_resolution_clock::now();
    // Calculate the elapsed time in milliseconds
    auto microsecondsCudaManaged = duration_cast<microseconds>(stopCudaManaged - startCudaManaged).count();


    
    // Ensure that the GPU has finished its work before proceeding
    hipDeviceSynchronize();

    // Print results
    std::cout << "Normal CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate (microsec): " << microsecondsCudaMalloc << std::endl;
    std::cout << "    Average distance from origin: " << avgDistanceCudaMalloc << std::endl;

    std::cout << "Pinned CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate (microsec): " << microsecondsCudaMallocHost << std::endl;
    std::cout << "    Average distance from origin: " << avgDistanceCudaMallocHost << std::endl;

    std::cout << "Managed CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate (microsec): " << microsecondsCudaManaged << std::endl;
    std::cout << "    Average distance from origin: " << avgDistanceCudaManaged << std::endl;
    std::cout << "Bye" << std::endl;

    // Free the individual rows first
    for (int i = 0; i < numWalkers; i++) {
        hipHostFree(states[i]);
    }

    // Then free the array of pointers
    hipHostFree(states);

    return 0;
}
