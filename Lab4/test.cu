#include <iostream>
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h> // CUDA Random Number Generation

// Initialize CUDA random number generator
__global__
void initRandomState(unsigned long long seed, hiprandState_t* states, int numWalkers)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < numWalkers)
    {
        hiprand_init(seed, id, 0, &states[id]);
    }
}

// Function to perform calculations using cudaMalloc
__global__
void calculateWithCudaMalloc(int numWalkers, int numSteps, float* distances, hiprandState_t* states)
{
    int walkerId = blockIdx.x * blockDim.x + threadIdx.x;
    if (walkerId < numWalkers)
    {
        int x = 0;
        int y = 0;
        hiprandState_t localState = states[walkerId]; // Get the random state for this walker
        for (int i = 0; i < numSteps; i++)
        {
            // Simulate a random step in 2D space
            int direction = hiprand(&localState) % 4;
            if (direction == 0) x++;
            else if (direction == 1) x--;
            else if (direction == 2) y++;
            else y--;
        }
        distances[walkerId] = sqrt(static_cast<float>(x * x + y * y));
    }
}

// Function to perform calculations using cudaMallocHost
void calculateWithCudaMallocHost(int numWalkers, int numSteps, float* distances)
{
    // Similar to the cudaMalloc version, but no need for device kernel
    for (int walkerId = 0; walkerId < numWalkers; walkerId++)
    {
        int x = 0;
        int y = 0;
        for (int i = 0; i < numSteps; i++)
        {
            // Simulate a random step in 2D space
            int direction = rand() % 4;
            if (direction == 0) x++;
            else if (direction == 1) x--;
            else if (direction == 2) y++;
            else y--;
        }
        distances[walkerId] = sqrt(static_cast<float>(x * x + y * y));
    }
}

// Function to perform calculations using cudaMallocManaged
__global__
void calculateWithCudaManaged(int numWalkers, int numSteps, float* distances, hiprandState_t* states)
{
    int walkerId = blockIdx.x * blockDim.x + threadIdx.x;
    if (walkerId < numWalkers)
    {
        int x = 0;
        int y = 0;
        hiprandState_t localState = states[walkerId]; // Get the random state for this walker
        for (int i = 0; i < numSteps; i++)
        {
            // Simulate a random step in 2D space
            int direction = hiprand(&localState) % 4;
            if (direction == 0) x++;
            else if (direction == 1) x--;
            else if (direction == 2) y++;
            else y--;
        }
        distances[walkerId] = sqrt(static_cast<float>(x * x + y * y));
    }
}

int main(int argc, char* argv[])
{
    if (argc != 5)
    {
        std::cerr << "Usage: " << argv[0] << " -W <Number of Walkers> -I <Number of Steps>" << std::endl;
        return 1;
    }

    int numWalkers = 0;
    int numSteps = 0;

    for (int i = 1; i < argc; i += 2)
    {
        std::string flag = argv[i];
        if (flag == "-W") numWalkers = std::atoi(argv[i + 1]);
        else if (flag == "-I") numSteps = std::atoi(argv[i + 1]);
    }

    if (numWalkers <= 0 || numSteps <= 0)
    {
        std::cerr << "Invalid input values. Number of walkers and steps must be positive integers." << std::endl;
        return 1;
    }

    // Allocate memory for distances using different methods
    float* distancesCudaMalloc = nullptr;
    float* distancesCudaMallocHost = nullptr;
    float* distancesCudaManaged = nullptr;
    hiprandState_t* randomStatesCudaMalloc = nullptr;
    hiprandState_t* randomStatesCudaManaged = nullptr;


    hipMalloc((void**)&distancesCudaMalloc, numWalkers * sizeof(float));
    hipHostMalloc((void**)&distancesCudaMallocHost, numWalkers * sizeof(float), hipHostMallocDefault);
    hipMallocManaged((void**)&distancesCudaManaged, numWalkers * sizeof(float));
    hipMalloc((void**)&randomStatesCudaMalloc, numWalkers * sizeof(hiprandState_t));
    hipMalloc((void**)&randomStatesCudaManaged, numWalkers * sizeof(hiprandState_t));

    int blockSize = 256;
    int numBlocks = (numWalkers + blockSize - 1) / blockSize;

    // Initialize random seed
    srand(static_cast<unsigned int>(time(nullptr)));
    initRandomState<<<numBlocks, blockSize>>>(time(nullptr), randomStatesCudaMalloc, numWalkers);
    initRandomState<<<numBlocks, blockSize>>>(time(nullptr), randomStatesCudaManaged, numWalkers);

    // Measure time for cudaMalloc
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    calculateWithCudaMalloc<<<numBlocks, blockSize>>>(numWalkers, numSteps, distancesCudaMalloc, randomStatesCudaMalloc);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float millisecondsCudaMalloc = 0;
    hipEventElapsedTime(&millisecondsCudaMalloc, start, stop);

    // Measure time for cudaMallocHost
    hipEventRecord(start);
    calculateWithCudaMallocHost(numWalkers, numSteps, distancesCudaMallocHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float millisecondsCudaMallocHost = 0;
    hipEventElapsedTime(&millisecondsCudaMallocHost, start, stop);

    // Measure time for cudaMallocManaged
    hipEventRecord(start);
    calculateWithCudaManaged<<<numBlocks, blockSize>>>(numWalkers, numSteps, distancesCudaManaged, randomStatesCudaManaged);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float millisecondsCudaManaged = 0;
    hipEventElapsedTime(&millisecondsCudaManaged, start, stop);

    // Copy data from GPU
    float* distancesCudaMallocCPU = nullptr;
    float* distancesCudaManagedCPU = nullptr;
    distancesCudaMallocCPU = (float*)malloc(numWalkers * sizeof(float));
    distancesCudaManagedCPU = (float*)malloc(numWalkers * sizeof(float));

    hipMemcpy(distancesCudaMalloc, distancesCudaMallocCPU, numWalkers * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(distancesCudaManaged, distancesCudaManagedCPU, numWalkers * sizeof(float), hipMemcpyDeviceToHost);


    // Calculate and print the average distance from the origin
    float avgDistanceCudaMalloc = 0.0f;
    float avgDistanceCudaMallocHost = 0.0f;
    float avgDistanceCudaManaged = 0.0f;

    for (int i = 0; i < numWalkers; i++)
    {
        avgDistanceCudaMalloc += distancesCudaMallocCPU[i];
        avgDistanceCudaMallocHost += distancesCudaMallocHost[i];
        avgDistanceCudaManaged += distancesCudaManagedCPU[i];
    }

    avgDistanceCudaMalloc /= static_cast<float>(numWalkers);
    avgDistanceCudaMallocHost /= static_cast<float>(numWalkers);
    avgDistanceCudaManaged /= static_cast<float>(numWalkers);

    // Print results
    std::cout << "Normal CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate (microsec): " << millisecondsCudaMalloc << std::endl;
    std::cout << "    Average distance from origin: " << avgDistanceCudaMalloc << std::endl;

    std::cout << "Pinned CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate (microsec): " << millisecondsCudaMallocHost << std::endl;
    std::cout << "    Average distance from origin: " << avgDistanceCudaMallocHost << std::endl;

    std::cout << "Managed CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate (microsec): " << millisecondsCudaManaged << std::endl;
    std::cout << "    Average distance from origin: " << avgDistanceCudaManaged << std::endl;
    std::cout << "Bye" << std::endl;

    // Clean up
    hipFree(distancesCudaMalloc);
    hipHostFree(distancesCudaMallocHost);
    hipFree(distancesCudaManaged);
    hipFree(randomStatesCudaMalloc);
    hipFree(randomStatesCudaManaged);
    free(distancesCudaMallocCPU);
    free(distancesCudaManagedCPU);

    return 0;
}
